#include "hip/hip_runtime.h"
/* 
 * Course: High Performance Computing 2021/2022
 * 
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Student:           
 * Canzolino	Gianluca  	06227001806  	g.canzolino3@studenti.unisa.it 
 *
 * Copyright (C) 2021 - All Rights Reserved
 *
 * This file is part of CommonAssignment1.
 *
 * CommonAssignment1 is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CommonAssignment1 is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CommonAssignment1.  If not, see <http://www.gnu.org/licenses/>.

 */

 /**
	@file BFS_texture.cu
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <unistd.h>
#include <math.h>

#define SIZE 300000
#define SIZE_NEIGHBOURS 1000
#define START_NODE 0
#define SIZE_FILENAME 50

#define STARTTIME \
  hipEvent_t start,stop;\
  hipEventCreate(&start);\
  hipEventCreate(&stop);\
  hipEventRecord(start,0);\

#define STOPTIME \
  hipEventRecord(stop,0);\
  hipEventSynchronize(stop);\
  float elapsed;\
  hipEventElapsedTime(&elapsed,start,stop);\
  elapsed/=1000.f;\
  hipEventDestroy(start);\
  hipEventDestroy(stop);\
  
#define PRINTTIME \
  printf("Kernel elapsed time %fs \n", elapsed);\
//

//Struct Graph
struct Graph {
  int numVertices;
  int max_neighbours;
  int* adjMatrix;
  int* visited;
  int* offset;
  int max_degree;
};

//Creazione del grafo
struct Graph* createGraph(int vertices) {
  struct Graph* graph = (struct Graph*) malloc(sizeof(struct Graph));
  graph->numVertices = vertices;
  int max_neig = 100;
  graph->max_neighbours = max_neig;

  graph->visited = (int *) malloc(vertices * sizeof(int));
  graph->adjMatrix = (int *) malloc(vertices * max_neig * sizeof(int));
  graph->offset = (int *) calloc(vertices, sizeof(int));

  for (int i = 0; i < vertices; i++) {
    graph->visited[i] = 0;
    for (int j = 0; j < max_neig; j++){
      graph->adjMatrix[i*max_neig + j] = -1;
    }
  }
  return graph;
}

//Aggiungi arco
void addEdge(struct Graph* graph, int src, int dest) {
  //Arco da src a dest
  graph->adjMatrix[src * graph->max_neighbours + graph->offset[src]++] = dest;
  //Arco da dest a src
  graph->adjMatrix[dest * graph->max_neighbours + graph->offset[dest]++] = src;
}

void make_csv(float time, int N, int gridsize, int thread_per_block){
  FILE* fp;
  char root_filename[] = "measure/texture";

  char* filename = (char*) malloc(sizeof(char) * (strlen(root_filename) + 10*sizeof(char)));
  sprintf(filename,"%s/%d/BFS_v_%d_b_%d_tpb.csv",root_filename,N,gridsize,thread_per_block);
  
  if ( access( filename, F_OK ) == 0 ) {
     fp = fopen(filename,"a"); 

  } else {
     fp = fopen(filename,"w");
     fprintf(fp, "N_vertices; BlockSize; GridSize; time_sec\n");
  }
  fprintf(fp, "%d; %d; %d; %f\n", N, thread_per_block, gridsize, time);
  fclose(fp);
}

texture<int,1> text_mem;

__global__ void computeNextQueue(int n_vertices, int *distance, int queueSize, int *currentQueue, int *nextQueueSize, int *nextQueue, int level) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;  // thread id
  //Creo un index per l'iterazione
  int idx_iter = tid;

	while (idx_iter < queueSize) {
		int current = currentQueue[idx_iter];
		for (int i = 0; i < n_vertices; i++) {
			int v = tex1Dfetch(text_mem,current * n_vertices + i);
			if(v==-1) break;
      if(distance[v] == 0){
        distance[v] = level + 1;
        int position = atomicAdd(nextQueueSize, 1);
        nextQueue[position] = v;
      }
		}
    idx_iter += blockDim.x * gridDim.x;
	}
}

int main(int argc,char ** argv){
  if(argc<4) {
    fprintf(stderr,"ERROR too few arguments of: %s\n",argv[0]);
    exit(1);
  }
  // Concateno il nome parziale dato in ingresso e ottengo due stringe:
  // 1) File dove è presente i dati per il grafo "graph.txt"
  // 2) File dove è presente la bfs di test per verificare il funzionamento "graph_test.txt" 
  char file_graph[SIZE_FILENAME];
  strcpy(file_graph, argv[1]);
  char file_test[SIZE_FILENAME] = ""; 
	strncat(file_test, file_graph, SIZE_FILENAME-1);
  strncat(file_test, "_test.txt", SIZE_FILENAME-1);
  strncat(file_graph, ".txt", SIZE_FILENAME-1);

  /*
  *
  * Inizio fase di lettura da file e creazione grafo
  *
  */
  // Apro il file di input
  FILE * fp = fopen(file_graph, "r");
  if (fp == NULL) exit(EXIT_FAILURE);

  int n_vertices = 0;
  char * line = NULL;
  size_t len = 0;
  if(getline(&line, &len, fp)!= -1){
    n_vertices = atoi(strtok(line, " "));
  }

  #ifdef DEBUG
    printf("Valori: %d\n", n_vertices);
  #endif

  struct Graph* graph = createGraph(n_vertices);
  char* other;
  int src_vertex;

  while (getline(&line, &len, fp) != -1) {
    src_vertex = atoi(strtok(line, " "));
    while(1) {
      other = strtok(NULL, " ");
      if(other == NULL){
        break;
      }
      addEdge(graph, src_vertex, atoi(other));
    }
  }
  fclose(fp);
  /*
  *
  * Fine fase di lettura da file e creazione grafo
  *
  */

  //Assegno il numero dei blocchi
  int blocks = atoi(argv[2]);
  dim3 dimGrid(blocks);

  //Assegno il numero dei thread per blocco
  int th_p_block = atoi(argv[3]);
  dim3 dimBlock(th_p_block);

  //Inizializzazioni variabili CPU
	int currentQueueSize = 1;
	const int NEXT_QUEUE_SIZE = 0;
  const int start_vertex = START_NODE;
	int level = 1;

  //Inizializzaione variabili GPU
  int *d_adjacencyMatrix;
	int *d_firstQueue;
	int *d_secondQueue;
	int *d_nextQueueSize;
  int *d_distance;

  const int size = n_vertices * sizeof(int);
  const int adjacencySize = n_vertices * graph->max_neighbours * sizeof(int);

  //Allocazione su GPU
  hipMalloc((void **)&d_adjacencyMatrix, adjacencySize);
  hipMalloc((void **)&d_firstQueue, size);
	hipMalloc((void **)&d_secondQueue, size);
  hipMalloc((void **)&d_distance, size);
	hipMalloc((void **)&d_nextQueueSize, sizeof(int));

  hipMemcpy(d_adjacencyMatrix, graph->adjMatrix, adjacencySize, hipMemcpyHostToDevice);
  hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_firstQueue, &start_vertex, sizeof(int), hipMemcpyHostToDevice);

  graph->visited[start_vertex] = 1;
  hipMemcpy(d_distance, graph->visited, n_vertices * sizeof(int), hipMemcpyHostToDevice);

  hipChannelFormatDesc channel = hipCreateChannelDesc<int>();
  hipBindTexture(0,text_mem, d_adjacencyMatrix, channel);

  //Avvio il timer
  STARTTIME
  while (currentQueueSize > 0) {
		int *d_currentQueue;
		int *d_nextQueue;
		if (level % 2 != 0) {
			d_currentQueue = d_firstQueue;
			d_nextQueue = d_secondQueue;
		}
		else {
			d_currentQueue = d_secondQueue;
			d_nextQueue = d_firstQueue;
		}
		computeNextQueue<<<blocks, th_p_block>>> (graph->max_neighbours, d_distance, currentQueueSize, d_currentQueue, d_nextQueueSize, d_nextQueue, level);
		hipDeviceSynchronize();
		level++;
		hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(d_nextQueueSize, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
	}
  //Salvo il tempo trascorso
  STOPTIME

  hipUnbindTexture(text_mem);

  hipMemcpy(graph->visited, d_distance, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  //Free delle variabili su GPU
  hipFree(d_adjacencyMatrix);
  hipFree(d_firstQueue);
  hipFree(d_secondQueue);
  hipFree(d_distance);

  /*
  *
  * Inizio fase di test per la correttezza della BFS
  *
  */
  FILE * fp_test;
  int work = 0;
  char* vertex;
  int actual_level = 0;

  fp_test = fopen(file_test, "r");
  if (fp_test == NULL) exit(EXIT_FAILURE);

  work = 1;
  actual_level = 0;

  while (getline(&line, &len, fp_test) != -1) {
    actual_level++;
    vertex = strtok(line, " ");
    if(graph->visited[atoi(vertex)] < actual_level){
      work = 0;
      break;
    }
    
    while(1) {
      vertex = strtok(NULL, " ");
      if(vertex == NULL){
        break;
      }
      if(graph->visited[atoi(vertex)] < actual_level){
        work = 0;
        break;
      }
    }

    if(work == 0){
      break;
    }
  }
  fclose(fp_test);

  /*
  *
  * Fine fase di test per la correttezza della BFS
  *
  */
    
  if(work == 1){
    PRINTTIME
    make_csv(elapsed, n_vertices, blocks, th_p_block);
    #ifdef DEBUG
      printf("\nWork!\n");
    #endif
  }
  
  return 0;
}